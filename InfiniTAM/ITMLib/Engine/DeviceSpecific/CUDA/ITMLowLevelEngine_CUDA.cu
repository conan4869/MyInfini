#include "hip/hip_runtime.h"
// Copyright 2014 Isis Innovation Limited and the authors of InfiniTAM

#include "ITMLowLevelEngine_CUDA.h"
#include "ITMCUDADefines.h"

#include "../../DeviceAgnostic/ITMLowLevelEngine.h"

using namespace ITMLib::Engine;

ITMLowLevelEngine_CUDA::ITMLowLevelEngine_CUDA(void) { }
ITMLowLevelEngine_CUDA::~ITMLowLevelEngine_CUDA(void) { }

__global__ void convertDisparityToDepth_device(float *depth_out, const short *depth_in, Vector2f disparityCalibParams, float fx_depth, Vector2i imgSize);

__global__ void filterSubsample_device(Vector4u *imageData_out, Vector2i newDims, const Vector4u *imageData_in, Vector2i oldDims);

__global__ void filterSubsampleWithHoles_device(float *imageData_out, Vector2i newDims, const float *imageData_in, Vector2i oldDims);
__global__ void filterSubsampleWithHoles_device(Vector4f *imageData_out, Vector2i newDims, const Vector4f *imageData_in, Vector2i oldDims);

__global__ void gradientX_device(Vector4s *grad, const Vector4u *image, Vector2i imgSize);
__global__ void gradientY_device(Vector4s *grad, const Vector4u *image, Vector2i imgSize);

// host methods

void ITMLowLevelEngine_CUDA::CopyImage(ITMUChar4Image *image_out, const ITMUChar4Image *image_in)
{
	Vector4u *dest = image_out->GetData(true);
	const Vector4u *src = image_in->GetData(true);

	ITMSafeCall(hipMemcpy(dest, src, image_in->dataSize * sizeof(Vector4u), hipMemcpyDeviceToDevice));
}

void ITMLowLevelEngine_CUDA::CopyImage(ITMFloatImage *image_out, const ITMFloatImage *image_in)
{
	float *dest = image_out->GetData(true);
	const float *src = image_in->GetData(true);

	ITMSafeCall(hipMemcpy(dest, src, image_in->dataSize * sizeof(float), hipMemcpyDeviceToDevice));
}

void ITMLowLevelEngine_CUDA::CopyImage(ITMFloat4Image *image_out, const ITMFloat4Image *image_in)
{
	Vector4f *dest = image_out->GetData(true);
	const Vector4f *src = image_in->GetData(true);

	ITMSafeCall(hipMemcpy(dest, src, image_in->dataSize * sizeof(Vector4f), hipMemcpyDeviceToDevice));
}

void ITMLowLevelEngine_CUDA::FilterSubsample(ITMUChar4Image *image_out, const ITMUChar4Image *image_in)
{
	Vector2i oldDims = image_in->noDims;
	Vector2i newDims; newDims.x = image_in->noDims.x / 2; newDims.y = image_in->noDims.y / 2;

	image_out->ChangeDims(newDims);

	const Vector4u *imageData_in = image_in->GetData(true);
	Vector4u *imageData_out = image_out->GetData(true);

	dim3 blockSize(16, 16);
	dim3 gridSize((int)ceil((float)newDims.x / (float)blockSize.x), (int)ceil((float)newDims.y / (float)blockSize.y));

	filterSubsample_device << <gridSize, blockSize >> >(imageData_out, newDims, imageData_in, oldDims);
}

void ITMLowLevelEngine_CUDA::FilterSubsampleWithHoles(ITMFloatImage *image_out, const ITMFloatImage *image_in)
{
	Vector2i oldDims = image_in->noDims;
	Vector2i newDims; newDims.x = image_in->noDims.x / 2; newDims.y = image_in->noDims.y / 2;

	image_out->ChangeDims(newDims);

	const float *imageData_in = image_in->GetData(true);
	float *imageData_out = image_out->GetData(true);

	dim3 blockSize(16, 16);
	dim3 gridSize((int)ceil((float)newDims.x / (float)blockSize.x), (int)ceil((float)newDims.y / (float)blockSize.y));

	filterSubsampleWithHoles_device << <gridSize, blockSize >> >(imageData_out, newDims, imageData_in, oldDims);
}

void ITMLowLevelEngine_CUDA::FilterSubsampleWithHoles(ITMFloat4Image *image_out, const ITMFloat4Image *image_in)
{
	Vector2i oldDims = image_in->noDims;
	Vector2i newDims; newDims.x = image_in->noDims.x / 2; newDims.y = image_in->noDims.y / 2;

	image_out->ChangeDims(newDims);

	const Vector4f *imageData_in = image_in->GetData(true);
	Vector4f *imageData_out = image_out->GetData(true);

	dim3 blockSize(16, 16);
	dim3 gridSize((int)ceil((float)newDims.x / (float)blockSize.x), (int)ceil((float)newDims.y / (float)blockSize.y));

	filterSubsampleWithHoles_device << <gridSize, blockSize >> >(imageData_out, newDims, imageData_in, oldDims);
}

void ITMLowLevelEngine_CUDA::GradientX(ITMShort4Image *grad_out, const ITMUChar4Image *image_in)
{
	grad_out->ChangeDims(image_in->noDims);
	Vector2i imgSize = image_in->noDims;

	Vector4s *grad = grad_out->GetData(true); 
	const Vector4u *image = image_in->GetData(true);

	dim3 blockSize(16, 16);
	dim3 gridSize((int)ceil((float)imgSize.x / (float)blockSize.x), (int)ceil((float)imgSize.y / (float)blockSize.y));

	ITMSafeCall(hipMemset(grad, 0, imgSize.x * imgSize.y * sizeof(Vector4s)));

	gradientX_device << <gridSize, blockSize >> >(grad, image, imgSize);
}

void ITMLowLevelEngine_CUDA::GradientY(ITMShort4Image *grad_out, const ITMUChar4Image *image_in)
{
	grad_out->ChangeDims(image_in->noDims);
	Vector2i imgSize = image_in->noDims;

	Vector4s *grad = grad_out->GetData(true);
	const Vector4u *image = image_in->GetData(true);

	dim3 blockSize(16, 16);
	dim3 gridSize((int)ceil((float)imgSize.x / (float)blockSize.x), (int)ceil((float)imgSize.y / (float)blockSize.y));

	ITMSafeCall(hipMemset(grad, 0, imgSize.x * imgSize.y * sizeof(Vector4s)));

	gradientY_device << <gridSize, blockSize >> >(grad, image, imgSize);
}

void ITMLowLevelEngine_CUDA::ConvertDisparityToDepth(ITMFloatImage *depth_out, const ITMShortImage *depth_in, const ITMIntrinsics *depthIntrinsics,
	const ITMDisparityCalib *disparityCalib)
{
	Vector2i imgSize = depth_in->noDims;

	const short *d_in = depth_in->GetData(true);
	float *d_out = depth_out->GetData(true);

	Vector2f disparityCalibParams; float fx_depth;
	disparityCalibParams.x = disparityCalib->params.x;
	disparityCalibParams.y = disparityCalib->params.y;
	fx_depth = depthIntrinsics->projectionParamsSimple.fx;

	dim3 blockSize(16, 16);
	dim3 gridSize((int)ceil((float)imgSize.x / (float)blockSize.x), (int)ceil((float)imgSize.y / (float)blockSize.y));

	convertDisparityToDepth_device << <gridSize, blockSize >> >(d_out, d_in, disparityCalibParams, fx_depth, imgSize);
}

// device functions

__global__ void convertDisparityToDepth_device(float *d_out, const short *d_in, Vector2f disparityCalibParams, float fx_depth, Vector2i imgSize)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	if ((x >= imgSize.x) || (y >= imgSize.y)) return;

	convertDisparityToDepth(d_out, x, y, d_in, disparityCalibParams, fx_depth, imgSize);
}

__global__ void filterSubsample_device(Vector4u *imageData_out, Vector2i newDims, const Vector4u *imageData_in, Vector2i oldDims)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x > newDims.x - 1 || y > newDims.y - 1) return;

	filterSubsample(imageData_out, x, y, newDims, imageData_in, oldDims);
}

__global__ void filterSubsampleWithHoles_device(float *imageData_out, Vector2i newDims, const float *imageData_in, Vector2i oldDims)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x > newDims.x - 1 || y > newDims.y - 1) return;

	filterSubsampleWithHoles(imageData_out, x, y, newDims, imageData_in, oldDims);
}

__global__ void filterSubsampleWithHoles_device(Vector4f *imageData_out, Vector2i newDims, const Vector4f *imageData_in, Vector2i oldDims)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x > newDims.x - 1 || y > newDims.y - 1) return;

	filterSubsampleWithHoles(imageData_out, x, y, newDims, imageData_in, oldDims);
}

__global__ void gradientX_device(Vector4s *grad, const Vector4u *image, Vector2i imgSize)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x < 2 || x > imgSize.x - 2 || y < 2 || y > imgSize.y - 2) return;

	gradientX(grad, x, y, image, imgSize);
}

__global__ void gradientY_device(Vector4s *grad, const Vector4u *image, Vector2i imgSize)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x < 2 || x > imgSize.x - 2 || y < 2 || y > imgSize.y - 2) return;

	gradientY(grad, x, y, image, imgSize);
}
