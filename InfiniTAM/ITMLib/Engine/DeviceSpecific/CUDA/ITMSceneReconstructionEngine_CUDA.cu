#include "hip/hip_runtime.h"
// Copyright 2014 Isis Innovation Limited and the authors of InfiniTAM

#include "ITMSceneReconstructionEngine_CUDA.h"
#include "ITMCUDAUtils.h"
#include "../../DeviceAgnostic/ITMSceneReconstructionEngine.h"

using namespace ITMLib::Engine;

template<class TVoxel>
__global__ void integrateIntoScene_device(TVoxel *localVBA, const ITMHashEntry *hashTable, int *noLiveEntryIDs, ITMHashCacheState *cacheStates,
	bool useSwapping, const Vector4u *rgb, Vector2i rgbImgSize, const float *depth, Vector2i imgSize, Matrix4f M_d, Matrix4f M_rgb, Vector4f projParams_d, 
	Vector4f projParams_rgb, float _voxelSize, float mu, int maxW);

template<class TVoxel>
__global__ void integrateIntoScene_device(TVoxel *voxelArray, const ITMPlainVoxelArray::ITMVoxelArrayInfo *arrayInfo,
	const Vector4u *rgb, Vector2i rgbImgSize, const float *depth, Vector2i depthImgSize, Matrix4f M_d, Matrix4f M_rgb, Vector4f projParams_d, 
	Vector4f projParams_rgb, float _voxelSize, float mu, int maxW);

__global__ void buildHashAllocAndVisibleType_device(uchar *entriesAllocType, uchar *entriesVisibleType, Vector3s *blockCoords, const float *depth,
	Matrix4f invM_d, Vector4f projParams_d, float mu, Vector2i _imgSize, float _voxelSize, ITMHashEntry *hashTable, float viewFrustum_min,
	float viewFrustrum_max);

__global__ void allocateVoxelBlocksList_device(int *voxelAllocationList, int *excessAllocationList, ITMHashEntry *hashTable, int noTotalEntries,
	int *noAllocatedVoxelEntries, int *noAllocatedExcessEntries, uchar *entriesAllocType, uchar *entriesVisibleType, Vector3s *blockCoords);

__global__ void reAllocateSwappedOutVoxelBlocks_device(int *voxelAllocationList, ITMHashEntry *hashTable, int noTotalEntries,
	int *noAllocatedVoxelEntries, uchar *entriesVisibleType);

__global__ void buildVisibleList_device(ITMHashEntry *hashTable, ITMHashCacheState *cacheStates, bool useSwapping, int noTotalEntries, 
	int *liveEntryIDs, int *noLiveEntries, uchar *entriesVisibleType, Matrix4f M_d, Vector4f projParams_d, Vector2i imgSize, float voxelSize);

// host methods

template<class TVoxel>
ITMSceneReconstructionEngine_CUDA<TVoxel,ITMVoxelBlockHash>::ITMSceneReconstructionEngine_CUDA(void) 
{
	ITMSafeCall(hipMalloc((void**)&noLiveEntries_device, sizeof(int)));
	ITMSafeCall(hipMalloc((void**)&noAllocatedVoxelEntries_device, sizeof(int)));
	ITMSafeCall(hipMalloc((void**)&noAllocatedExcessEntries_device, sizeof(int)));

	int noTotalEntries = ITMVoxelBlockHash::noVoxelBlocks;
	ITMSafeCall(hipMalloc((void**)&entriesAllocType_device, noTotalEntries));
	ITMSafeCall(hipMalloc((void**)&blockCoords_device, noTotalEntries * sizeof(Vector3s)));
}

template<class TVoxel>
ITMSceneReconstructionEngine_CUDA<TVoxel,ITMVoxelBlockHash>::~ITMSceneReconstructionEngine_CUDA(void) 
{
	ITMSafeCall(hipFree(noLiveEntries_device));
	ITMSafeCall(hipFree(noAllocatedVoxelEntries_device));
	ITMSafeCall(hipFree(noAllocatedExcessEntries_device));

	ITMSafeCall(hipFree(entriesAllocType_device));
	ITMSafeCall(hipFree(blockCoords_device));
}

template<class TVoxel>
void ITMSceneReconstructionEngine_CUDA<TVoxel,ITMVoxelBlockHash>::AllocateSceneFromDepth(ITMScene<TVoxel,ITMVoxelBlockHash> *scene, const ITMView *view, const ITMPose *pose_d)
{
	Vector2i depthImgSize = view->depth->noDims;
	float voxelSize = scene->sceneParams->voxelSize;

	Matrix4f M_d, invM_d;
	Vector4f projParams_d, invProjParams_d;

	M_d = pose_d->M; M_d.inv(invM_d);

	projParams_d = view->calib->intrinsics_d.projectionParamsSimple.all;
	invProjParams_d = projParams_d;
	invProjParams_d.x = 1.0f / invProjParams_d.x;
	invProjParams_d.y = 1.0f / invProjParams_d.y;

	float mu = scene->sceneParams->mu;

	float *depth = view->depth->GetData(true);
	int *voxelAllocationList = scene->localVBA.GetAllocationList();
	int *excessAllocationList = scene->index.GetExcessAllocationList();
	uchar *entriesVisibleType = scene->index.GetEntriesVisibleType();
	ITMHashEntry *hashTable = scene->index.GetEntries();
	ITMHashCacheState *cacheStates = scene->useSwapping ? scene->globalCache->GetCacheStates(true) : 0;
	int *liveEntryIDs = scene->index.GetLiveEntryIDs();
	int noTotalEntries = scene->index.noVoxelBlocks;

	float oneOverVoxelSize = 1.0f / (voxelSize * SDF_BLOCK_SIZE);

	dim3 cudaBlockSizeHV(16, 16);
	dim3 gridSizeHV((int)ceil((float)depthImgSize.x / (float)cudaBlockSizeHV.x), (int)ceil((float)depthImgSize.y / (float)cudaBlockSizeHV.y));

	ITMSafeCall(hipMemcpy(noAllocatedVoxelEntries_device, &scene->localVBA.lastFreeBlockId, sizeof(int), hipMemcpyHostToDevice));
	ITMSafeCall(hipMemcpy(noAllocatedExcessEntries_device, &scene->index.lastFreeExcessListId, sizeof(int), hipMemcpyHostToDevice));
	ITMSafeCall(hipMemset(noLiveEntries_device, 0, sizeof(int)));

	ITMSafeCall(hipMemset(entriesAllocType_device, 0, sizeof(unsigned char)* noTotalEntries));
	ITMSafeCall(hipMemset(entriesVisibleType, 0, sizeof(unsigned char)* noTotalEntries));
	ITMSafeCall(hipMemset(blockCoords_device, 0, sizeof(Vector3s)* noTotalEntries));

	buildHashAllocAndVisibleType_device << <gridSizeHV, cudaBlockSizeHV >> >(entriesAllocType_device, entriesVisibleType, 
		blockCoords_device, depth, invM_d, invProjParams_d, mu, depthImgSize, oneOverVoxelSize, hashTable,
		scene->sceneParams->viewFrustum_min, scene->sceneParams->viewFrustum_max);

	dim3 cudaBlockSizeAL(256, 1);
	dim3 gridSizeAL((int)ceil((float)noTotalEntries / (float)cudaBlockSizeAL.x));

	allocateVoxelBlocksList_device << <gridSizeAL, cudaBlockSizeAL >> >(voxelAllocationList, excessAllocationList, hashTable,
		noTotalEntries, noAllocatedVoxelEntries_device, noAllocatedExcessEntries_device, entriesAllocType_device, entriesVisibleType, 
		blockCoords_device);

	buildVisibleList_device << <gridSizeAL, cudaBlockSizeAL >> >(hashTable, cacheStates, scene->useSwapping, noTotalEntries, liveEntryIDs,
		noLiveEntries_device, entriesVisibleType, M_d, projParams_d, depthImgSize, voxelSize);

	if (scene->useSwapping)
	{
		reAllocateSwappedOutVoxelBlocks_device << <gridSizeAL, cudaBlockSizeAL >> >(voxelAllocationList, hashTable, noTotalEntries, 
			noAllocatedVoxelEntries_device, entriesVisibleType);
	}

	ITMSafeCall(hipMemcpy(&scene->index.noLiveEntries, noLiveEntries_device, sizeof(int), hipMemcpyDeviceToHost));
	ITMSafeCall(hipMemcpy(&scene->localVBA.lastFreeBlockId, noAllocatedVoxelEntries_device, sizeof(int), hipMemcpyDeviceToHost));
	ITMSafeCall(hipMemcpy(&scene->index.lastFreeExcessListId, noAllocatedExcessEntries_device, sizeof(int), hipMemcpyDeviceToHost));
}

template<class TVoxel>
void ITMSceneReconstructionEngine_CUDA<TVoxel,ITMVoxelBlockHash>::IntegrateIntoScene(ITMScene<TVoxel,ITMVoxelBlockHash> *scene, const ITMView *view, const ITMPose *pose_d)
{
	Vector2i rgbImgSize = view->rgb->noDims;
	Vector2i depthImgSize = view->depth->noDims;
	float voxelSize = scene->sceneParams->voxelSize;

	Matrix4f M_d, M_rgb;
	Vector4f projParams_d, projParams_rgb;

	M_d = pose_d->M;
	if (TVoxel::hasColorInformation) M_rgb = view->calib->trafo_rgb_to_depth.calib_inv * pose_d->M;

	projParams_d = view->calib->intrinsics_d.projectionParamsSimple.all;
	projParams_rgb = view->calib->intrinsics_rgb.projectionParamsSimple.all;

	float mu = scene->sceneParams->mu; int maxW = scene->sceneParams->maxW;

	float *depth = view->depth->GetData(true);
	Vector4u *rgb = view->rgb->GetData(true);
	TVoxel *localVBA = scene->localVBA.GetVoxelBlocks();
	ITMHashEntry *hashTable = scene->index.GetEntries();
	ITMHashCacheState *cacheStates = scene->useSwapping ? scene->globalCache->GetCacheStates(true) : 0;
	int *liveEntryIDs = scene->index.GetLiveEntryIDs();

	dim3 cudaBlockSize(SDF_BLOCK_SIZE, SDF_BLOCK_SIZE, SDF_BLOCK_SIZE);
	dim3 gridSize(scene->index.noLiveEntries);

	integrateIntoScene_device << <gridSize, cudaBlockSize >> >(localVBA, hashTable, liveEntryIDs, cacheStates,
		scene->useSwapping, rgb, rgbImgSize, depth, depthImgSize, M_d, M_rgb, projParams_d, projParams_rgb, voxelSize, mu, maxW);
}

// plain voxel array

template<class TVoxel>
void ITMSceneReconstructionEngine_CUDA<TVoxel,ITMPlainVoxelArray>::AllocateSceneFromDepth(ITMScene<TVoxel,ITMPlainVoxelArray> *scene, const ITMView *view, const ITMPose *pose_d)
{
}

template<class TVoxel>
void ITMSceneReconstructionEngine_CUDA<TVoxel,ITMPlainVoxelArray>::IntegrateIntoScene(ITMScene<TVoxel,ITMPlainVoxelArray> *scene, const ITMView *view, const ITMPose *pose_d)
{
	Vector2i rgbImgSize = view->rgb->noDims;
	Vector2i depthImgSize = view->depth->noDims;
	float voxelSize = scene->sceneParams->voxelSize;

	Matrix4f M_d, M_rgb;
	Vector4f projParams_d, projParams_rgb;

	M_d = pose_d->M;
	if (TVoxel::hasColorInformation) M_rgb = view->calib->trafo_rgb_to_depth.calib_inv * pose_d->M;

	projParams_d = view->calib->intrinsics_d.projectionParamsSimple.all;
	projParams_rgb = view->calib->intrinsics_rgb.projectionParamsSimple.all;

	float mu = scene->sceneParams->mu; int maxW = scene->sceneParams->maxW;

	float *depth = view->depth->GetData(true);
	Vector4u *rgb = view->rgb->GetData(true);
	TVoxel *localVBA = scene->localVBA.GetVoxelBlocks();
	const ITMPlainVoxelArray::ITMVoxelArrayInfo *arrayInfo = scene->index.getIndexData();

	dim3 cudaBlockSize(8, 8, 8);
	dim3 gridSize(scene->index.getVolumeSize().x / cudaBlockSize.x, scene->index.getVolumeSize().y / cudaBlockSize.y, scene->index.getVolumeSize().z / cudaBlockSize.z);

	integrateIntoScene_device << <gridSize, cudaBlockSize >> >(localVBA, arrayInfo,
		rgb, rgbImgSize, depth, depthImgSize, M_d, M_rgb, projParams_d, projParams_rgb, voxelSize, mu, maxW);
}

// device functions

template<class TVoxel>
__global__ void integrateIntoScene_device(TVoxel *voxelArray, const ITMPlainVoxelArray::ITMVoxelArrayInfo *arrayInfo,
	const Vector4u *rgb, Vector2i rgbImgSize, const float *depth, Vector2i depthImgSize, Matrix4f M_d, Matrix4f M_rgb, Vector4f projParams_d, 
	Vector4f projParams_rgb, float _voxelSize, float mu, int maxW)
{
	int x = blockIdx.x*blockDim.x+threadIdx.x;
	int y = blockIdx.y*blockDim.y+threadIdx.y;
	int z = blockIdx.z*blockDim.z+threadIdx.z;

	Vector4f pt_model; int locId;

	locId = x + y * arrayInfo->size.x + z * arrayInfo->size.x * arrayInfo->size.y;

	pt_model.x = (float)(x + arrayInfo->offset.x) * _voxelSize;
	pt_model.y = (float)(y + arrayInfo->offset.y) * _voxelSize;
	pt_model.z = (float)(z + arrayInfo->offset.z) * _voxelSize;
	pt_model.w = 1.0f;

	ComputeUpdatedVoxelInfo<TVoxel::hasColorInformation,TVoxel>::compute(voxelArray[locId], pt_model, M_d, projParams_d, M_rgb, projParams_rgb, mu, maxW, depth, depthImgSize, rgb, rgbImgSize);
}

template<class TVoxel>
__global__ void integrateIntoScene_device(TVoxel *localVBA, const ITMHashEntry *hashTable, int *liveEntryIDs, ITMHashCacheState *cacheStates,
	bool useSwapping, const Vector4u *rgb, Vector2i rgbImgSize, const float *depth, Vector2i depthImgSize, Matrix4f M_d, Matrix4f M_rgb, Vector4f projParams_d, 
	Vector4f projParams_rgb, float _voxelSize, float mu, int maxW)
{
	Vector3i globalPos;
	int entryId = liveEntryIDs[blockIdx.x];

	const ITMHashEntry &currentHashEntry = hashTable[entryId];

	if (currentHashEntry.ptr < 0) return;

	globalPos = currentHashEntry.pos.toInt() * SDF_BLOCK_SIZE;

	TVoxel *localVoxelBlock = &(localVBA[currentHashEntry.ptr * SDF_BLOCK_SIZE3]);

	int x = threadIdx.x, y = threadIdx.y, z = threadIdx.z;

	Vector4f pt_model; int locId;

	locId = x + y * SDF_BLOCK_SIZE + z * SDF_BLOCK_SIZE * SDF_BLOCK_SIZE;

	pt_model.x = (float)(globalPos.x + x) * _voxelSize;
	pt_model.y = (float)(globalPos.y + y) * _voxelSize;
	pt_model.z = (float)(globalPos.z + z) * _voxelSize;
	pt_model.w = 1.0f;

	ComputeUpdatedVoxelInfo<TVoxel::hasColorInformation,TVoxel>::compute(localVoxelBlock[locId], pt_model, M_d, projParams_d, M_rgb, projParams_rgb, mu, maxW, depth, depthImgSize, rgb, rgbImgSize);
}

__global__ void buildHashAllocAndVisibleType_device(uchar *entriesAllocType, uchar *entriesVisibleType, Vector3s *blockCoords, const float *depth,
	Matrix4f invM_d, Vector4f projParams_d, float mu, Vector2i _imgSize, float _voxelSize, ITMHashEntry *hashTable, float viewFrustum_min,
	float viewFrustum_max)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x > _imgSize.x - 1 || y > _imgSize.y - 1) return;

	buildHashAllocAndVisibleTypePP(entriesAllocType, entriesVisibleType, x, y, blockCoords, depth, invM_d,
		projParams_d, mu, _imgSize, _voxelSize, hashTable, viewFrustum_min, viewFrustum_max);
}

__global__ void allocateVoxelBlocksList_device(int *voxelAllocationList, int *excessAllocationList, ITMHashEntry *hashTable, int noTotalEntries,
	int *noAllocatedVoxelEntries, int *noAllocatedExcessEntries, uchar *entriesAllocType, uchar *entriesVisibleType, Vector3s *blockCoords)
{
	int targetIdx = threadIdx.x + blockIdx.x * blockDim.x;
	if (targetIdx > noTotalEntries - 1) return;

	int vbaIdx, exlIdx;
	ITMHashEntry hashEntry = hashTable[targetIdx];

	switch (entriesAllocType[targetIdx])
	{
	case 1: //needs allocation, fits in the ordered list
		vbaIdx = atomicSub(&noAllocatedVoxelEntries[0], 1);

		if (vbaIdx >= 0) //there is room in the voxel block array
		{
			Vector3s pt_block_all = blockCoords[targetIdx];

			hashEntry.pos.x = pt_block_all.x; hashEntry.pos.y = pt_block_all.y; hashEntry.pos.z = pt_block_all.z;
			hashEntry.ptr = voxelAllocationList[vbaIdx];

			hashTable[targetIdx] = hashEntry;
		}
		break;

	case 2: //needs allocation in the excess list
		vbaIdx = atomicSub(&noAllocatedVoxelEntries[0], 1);
		exlIdx = atomicSub(&noAllocatedExcessEntries[0], 1);

		if (vbaIdx >= 0 && exlIdx >= 0) //there is room in the voxel block array and excess list
		{
			Vector3s pt_block_all = blockCoords[targetIdx];

			hashEntry.pos.x = pt_block_all.x; hashEntry.pos.y = pt_block_all.y; hashEntry.pos.z = pt_block_all.z;
			hashEntry.ptr = voxelAllocationList[vbaIdx];

			int exlOffset = excessAllocationList[exlIdx];

			hashTable[targetIdx].offset = exlOffset + 1; //connect to child

			hashTable[SDF_BUCKET_NUM * SDF_ENTRY_NUM_PER_BUCKET + exlOffset] = hashEntry; //add child to the excess list

			entriesVisibleType[SDF_BUCKET_NUM * SDF_ENTRY_NUM_PER_BUCKET + exlOffset] = 1; //make child visible
		}

		break;
	}
}

__global__ void reAllocateSwappedOutVoxelBlocks_device(int *voxelAllocationList, ITMHashEntry *hashTable, int noTotalEntries,
	int *noAllocatedVoxelEntries, uchar *entriesVisibleType)
{
	int targetIdx = threadIdx.x + blockIdx.x * blockDim.x;
	if (targetIdx > noTotalEntries - 1) return;

	int vbaIdx;
	ITMHashEntry hashEntry = hashTable[targetIdx];

	if (entriesVisibleType[targetIdx] > 0 && hashEntry.ptr == -1) //it is visible and has been previously allocated inside the hash, but deallocated from VBA
	{
		vbaIdx = atomicSub(&noAllocatedVoxelEntries[0], 1);
		if (vbaIdx >= 0) hashTable[targetIdx].ptr = voxelAllocationList[vbaIdx];
	}
}

__global__ void buildVisibleList_device(ITMHashEntry *hashTable, ITMHashCacheState *cacheStates, bool useSwapping, int noTotalEntries, 
	int *liveEntryIDs, int *noLiveEntries, uchar *entriesVisibleType, Matrix4f M_d, Vector4f projParams_d, Vector2i imgSize, float voxelSize)
{
	int targetIdx = threadIdx.x + blockIdx.x * blockDim.x;
	if (targetIdx > noTotalEntries - 1) return;

	__shared__ bool shouldPrefix;

	unsigned char hashVisibleType = entriesVisibleType[targetIdx];
	const ITMHashEntry &hashEntry = hashTable[targetIdx];

	shouldPrefix = false;
	__syncthreads();

	if (hashVisibleType > 0) shouldPrefix = true;
	else
	{
		if (hashEntry.ptr >= -1)
		{
			shouldPrefix = true;

			Vector3f pt_image, buff3f;

			int noInvisible = 0, noInvisibleEnlarged = 0;

			pt_image = hashEntry.pos.toFloat() * (float)SDF_BLOCK_SIZE * voxelSize;
			buff3f = M_d * pt_image;

			if (buff3f.z > 1e-10f)
			{
				shouldPrefix = true;

				for (int x = 0; x <= 1; x++) for (int y = 0; y <= 1; y++) for (int z = 0; z <= 1; z++)
				{
					Vector3f off((float)x, (float)y, (float)z);

					pt_image = (hashEntry.pos.toFloat() + off) * (float)SDF_BLOCK_SIZE * voxelSize;

					buff3f = M_d * pt_image;

					pt_image.x = projParams_d.x * buff3f.x / buff3f.z + projParams_d.z;
					pt_image.y = projParams_d.y * buff3f.y / buff3f.z + projParams_d.w;

					if (!(pt_image.x >= 0 && pt_image.x < imgSize.x && pt_image.y >= 0 && pt_image.y < imgSize.y)) noInvisible++;

					if (useSwapping)
					{
						Vector4i lims;
						lims.x = -imgSize.x / 8; lims.y = imgSize.x + imgSize.x / 8;
						lims.z = -imgSize.y / 8; lims.w = imgSize.y + imgSize.y / 8;

						if (!(pt_image.x >= lims.x && pt_image.x < lims.y && pt_image.y >= lims.z && pt_image.y < lims.w)) noInvisibleEnlarged++;
					}
				}

				hashVisibleType = noInvisible < 8;

				if (useSwapping) entriesVisibleType[targetIdx] = noInvisibleEnlarged < 8;
			}
		}
	}

	if (useSwapping)
	{
		if (entriesVisibleType[targetIdx] > 0 && cacheStates[targetIdx].cacheFromHost != 2) cacheStates[targetIdx].cacheFromHost = 1;
	}

	__syncthreads();

	if (shouldPrefix)
	{
		int offset = computePrefixSum_device<int>(hashVisibleType > 0, noLiveEntries, blockDim.x * blockDim.y, threadIdx.x);
		if (offset != -1) liveEntryIDs[offset] = targetIdx;
	}
}

template class ITMSceneReconstructionEngine_CUDA<ITMVoxel,ITMVoxelIndex>;

