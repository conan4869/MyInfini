#include "hip/hip_runtime.h"
// Copyright 2014 Isis Innovation Limited and the authors of InfiniTAM

#include "ITMSwappingEngine_CUDA.h"
#include "ITMCUDAUtils.h"
#include "../../DeviceAgnostic/ITMSwappingEngine.h"

using namespace ITMLib::Engine;

__global__ void buildNeededListFromHost_device(int *neededEntryIDs, int *noNeededEntries, ITMHashCacheState *cacheStates, int noTotalEntries);

template<class TVoxel>
__global__ void integrateGlobalIntoLocal_device(TVoxel *localVBA, ITMHashCacheState *cacheStates, TVoxel *syncedVoxelBlocks_local,
	int *neededEntryIDs_local, ITMHashEntry *hashTable, int maxW);

__global__ void buildNeededListToHost_device(int *neededEntryIDs, int *noNeededEntries, ITMHashCacheState *cacheStates,
	ITMHashEntry *hashTable, uchar *entriesVisibleType, int noTotalEntries);

template<class TVoxel>
__global__ void cleanMemory_device(int *voxelAllocationList, int *noAllocatedVoxelEntries, ITMHashCacheState *cacheStates,
	ITMHashEntry *hashTable, TVoxel *localVBA, int *neededEntryIDs_local, int noNeededEntries);

template<class TVoxel>
__global__ void buildLocalSyncedStorageToHost_device(TVoxel *syncedVoxelBlocks_local, bool *hasSyncedData_local,
	int *neededEntryIDs_local, ITMHashEntry *hashTable, TVoxel *localVBA);

template<class TVoxel>
__global__ void cleanVBA(TVoxel *localVBA, int *voxelAllocationList, int baseAddressToClean);

template<class TVoxel>
ITMSwappingEngine_CUDA<TVoxel,ITMVoxelBlockHash>::ITMSwappingEngine_CUDA(void)
{
	ITMSafeCall(hipMalloc((void**)&noAllocatedVoxelEntries_device, sizeof(int)));
	ITMSafeCall(hipMalloc((void**)&noNeededEntries_device, sizeof(int)));
}

template<class TVoxel>
ITMSwappingEngine_CUDA<TVoxel,ITMVoxelBlockHash>::~ITMSwappingEngine_CUDA(void)
{
	ITMSafeCall(hipFree(noAllocatedVoxelEntries_device));
	ITMSafeCall(hipFree(noNeededEntries_device));
}

template<class TVoxel>
int ITMSwappingEngine_CUDA<TVoxel,ITMVoxelBlockHash>::DownloadFromGlobalMemory(ITMScene<TVoxel,ITMVoxelBlockHash> *scene, ITMView *view)
{
	ITMGlobalCache<TVoxel> *globalCache = scene->globalCache;

	ITMHashCacheState *cacheStates = globalCache->GetCacheStates(true);

	TVoxel *syncedVoxelBlocks_local = globalCache->GetSyncedVoxelBlocks(true);
	bool *hasSyncedData_local = globalCache->GetHasSyncedData(true);
	int *neededEntryIDs_local = globalCache->GetNeededEntryIDs(true);

	TVoxel *syncedVoxelBlocks_global = globalCache->GetSyncedVoxelBlocks(false);
	bool *hasSyncedData_global = globalCache->GetHasSyncedData(false);
	int *neededEntryIDs_global = globalCache->GetNeededEntryIDs(false);

	dim3 blockSize(256);
	dim3 gridSize((int)ceil((float)scene->index.noVoxelBlocks / (float)blockSize.x));

	ITMSafeCall(hipMemset(noNeededEntries_device, 0, sizeof(int)));

	buildNeededListFromHost_device << <gridSize, blockSize >> >(neededEntryIDs_local, noNeededEntries_device, cacheStates,
		scene->globalCache->noTotalEntries);

	int noNeededEntries;
	ITMSafeCall(hipMemcpy(&noNeededEntries, noNeededEntries_device, sizeof(int), hipMemcpyDeviceToHost));

	if (noNeededEntries > 0)
	{
		noNeededEntries = MIN(noNeededEntries, SDF_TRANSFER_BLOCK_NUM);
		ITMSafeCall(hipMemcpy(neededEntryIDs_global, neededEntryIDs_local, sizeof(int) * noNeededEntries, hipMemcpyDeviceToHost));

		memset(syncedVoxelBlocks_global, 0, noNeededEntries * SDF_BLOCK_SIZE3 * sizeof(TVoxel));
		memset(hasSyncedData_global, 0, noNeededEntries * sizeof(bool));
		for (int i = 0; i < noNeededEntries; i++)
		{
			int entryId = neededEntryIDs_global[i];

			if (globalCache->HasStoredData(entryId))
			{
				hasSyncedData_global[i] = true;
				memcpy(syncedVoxelBlocks_global + i * SDF_BLOCK_SIZE3, globalCache->GetStoredVoxelBlock(entryId), SDF_BLOCK_SIZE3 * sizeof(TVoxel));
			}
		}

		ITMSafeCall(hipMemcpy(hasSyncedData_local, hasSyncedData_global, sizeof(bool) * noNeededEntries, hipMemcpyHostToDevice));
		ITMSafeCall(hipMemcpy(syncedVoxelBlocks_local, syncedVoxelBlocks_global, sizeof(TVoxel) *SDF_BLOCK_SIZE3 * noNeededEntries, hipMemcpyHostToDevice));
	}

	return noNeededEntries;
}

template<class TVoxel>
void ITMSwappingEngine_CUDA<TVoxel,ITMVoxelBlockHash>::IntegrateGlobalIntoLocal(ITMScene<TVoxel,ITMVoxelBlockHash> *scene, ITMView *view)
{
	ITMGlobalCache<TVoxel> *globalCache = scene->globalCache;

	ITMHashEntry *hashTable = scene->index.GetEntries();

	ITMHashCacheState *cacheStates = globalCache->GetCacheStates(true);

	TVoxel *syncedVoxelBlocks_local = globalCache->GetSyncedVoxelBlocks(true);
	int *neededEntryIDs_local = globalCache->GetNeededEntryIDs(true);

	TVoxel *localVBA = scene->localVBA.GetVoxelBlocks();

	int noNeededEntries = this->DownloadFromGlobalMemory(scene, view);

	int maxW = scene->sceneParams->maxW;

	dim3 blockSize(SDF_BLOCK_SIZE, SDF_BLOCK_SIZE, SDF_BLOCK_SIZE);
	dim3 gridSize(noNeededEntries);

	integrateGlobalIntoLocal_device << <gridSize, blockSize >> >(localVBA, cacheStates, syncedVoxelBlocks_local,
		neededEntryIDs_local, hashTable, maxW);
}

template<class TVoxel>
void ITMSwappingEngine_CUDA<TVoxel,ITMVoxelBlockHash>::SaveToGlobalMemory(ITMScene<TVoxel,ITMVoxelBlockHash> *scene, ITMView *view)
{
	ITMGlobalCache<TVoxel> *globalCache = scene->globalCache;

	ITMHashCacheState *cacheStates = globalCache->GetCacheStates(true);

	ITMHashEntry *hashTable = scene->index.GetEntries();
	uchar *entriesVisibleType = scene->index.GetEntriesVisibleType();
	
	TVoxel *syncedVoxelBlocks_local = globalCache->GetSyncedVoxelBlocks(true);
	bool *hasSyncedData_local = globalCache->GetHasSyncedData(true);
	int *neededEntryIDs_local = globalCache->GetNeededEntryIDs(true);

	TVoxel *syncedVoxelBlocks_global = globalCache->GetSyncedVoxelBlocks(false);
	bool *hasSyncedData_global = globalCache->GetHasSyncedData(false);
	int *neededEntryIDs_global = globalCache->GetNeededEntryIDs(false);

	TVoxel *localVBA = scene->localVBA.GetVoxelBlocks();
	int *voxelAllocationList = scene->localVBA.GetAllocationList();

	int noTotalEntries = globalCache->noTotalEntries;
	
	dim3 blockSize, gridSize;
	int noNeededEntries;

	{
		blockSize = dim3(256);
		gridSize = dim3((int)ceil((float)scene->index.noVoxelBlocks / (float)blockSize.x));

		ITMSafeCall(hipMemset(noNeededEntries_device, 0, sizeof(int)));

		buildNeededListToHost_device << <gridSize, blockSize >> >(neededEntryIDs_local, noNeededEntries_device, cacheStates,
			hashTable, entriesVisibleType, noTotalEntries);

		ITMSafeCall(hipMemcpy(&noNeededEntries, noNeededEntries_device, sizeof(int), hipMemcpyDeviceToHost));
	}

	if (noNeededEntries > 0)
	{
		noNeededEntries = MIN(noNeededEntries, SDF_TRANSFER_BLOCK_NUM);
		{
			blockSize = dim3(SDF_BLOCK_SIZE, SDF_BLOCK_SIZE, SDF_BLOCK_SIZE);
			gridSize = dim3(noNeededEntries);

			buildLocalSyncedStorageToHost_device << <gridSize, blockSize >> >(syncedVoxelBlocks_local, hasSyncedData_local,
				neededEntryIDs_local, hashTable, localVBA);
		}

		int noBeforeCleanup = scene->localVBA.lastFreeBlockId;

		{
			blockSize = dim3(256);
			gridSize = dim3((int)ceil((float)noNeededEntries / (float)blockSize.x));

			ITMSafeCall(hipMemcpy(noAllocatedVoxelEntries_device, &scene->localVBA.lastFreeBlockId, sizeof(int), hipMemcpyHostToDevice));

			cleanMemory_device << <gridSize, blockSize >> >(voxelAllocationList, noAllocatedVoxelEntries_device, cacheStates, hashTable, localVBA,
				neededEntryIDs_local, noNeededEntries);

			ITMSafeCall(hipMemcpy(&scene->localVBA.lastFreeBlockId, noAllocatedVoxelEntries_device, sizeof(int), hipMemcpyDeviceToHost));
			scene->localVBA.lastFreeBlockId = MAX(scene->localVBA.lastFreeBlockId, 0);
			scene->localVBA.lastFreeBlockId = MIN(scene->localVBA.lastFreeBlockId, SDF_LOCAL_BLOCK_NUM);
		}

		if (scene->localVBA.lastFreeBlockId > noBeforeCleanup && noBeforeCleanup > 0)
		{
			blockSize = dim3(SDF_BLOCK_SIZE, SDF_BLOCK_SIZE, SDF_BLOCK_SIZE);
			gridSize = dim3(scene->localVBA.lastFreeBlockId - noBeforeCleanup);

			cleanVBA << <gridSize, blockSize >> >(localVBA, voxelAllocationList, noBeforeCleanup + 1);
		}

		ITMSafeCall(hipMemcpy(neededEntryIDs_global, neededEntryIDs_local, sizeof(int) * noNeededEntries, hipMemcpyDeviceToHost));
		ITMSafeCall(hipMemcpy(hasSyncedData_global, hasSyncedData_local, sizeof(bool) * noNeededEntries, hipMemcpyDeviceToHost));
		ITMSafeCall(hipMemcpy(syncedVoxelBlocks_global, syncedVoxelBlocks_local, sizeof(TVoxel) *SDF_BLOCK_SIZE3 * noNeededEntries, hipMemcpyDeviceToHost));

		for (int entryId = 0; entryId < noNeededEntries; entryId++)
		{
			if (hasSyncedData_global[entryId])
				globalCache->SetStoredData(neededEntryIDs_global[entryId], syncedVoxelBlocks_global + entryId * SDF_BLOCK_SIZE3);
		}
	}
}

__global__ void buildNeededListFromHost_device(int *neededEntryIDs, int *noNeededEntries, ITMHashCacheState *cacheStates, int noTotalEntries)
{
	int targetIdx = threadIdx.x + blockIdx.x * blockDim.x;
	if (targetIdx > noTotalEntries - 1) return;

	__shared__ bool shouldPrefix;

	shouldPrefix = false;
	__syncthreads();

	bool isNeededId = (cacheStates[targetIdx].cacheFromHost == 1);

	if (isNeededId) shouldPrefix = true;
	__syncthreads();

	if (shouldPrefix)
	{
		int offset = computePrefixSum_device<int>(isNeededId, noNeededEntries, blockDim.x * blockDim.y, threadIdx.x);
		if (offset != -1 && offset < SDF_TRANSFER_BLOCK_NUM) neededEntryIDs[offset] = targetIdx;
	}
}

__global__ void buildNeededListToHost_device(int *neededEntryIDs, int *noNeededEntries, ITMHashCacheState *cacheStates, 
	ITMHashEntry *hashTable, uchar *entriesVisibleType, int noTotalEntries)
{
	int targetIdx = threadIdx.x + blockIdx.x * blockDim.x;
	if (targetIdx > noTotalEntries - 1) return;

	__shared__ bool shouldPrefix;

	shouldPrefix = false;
	__syncthreads();

	ITMHashCacheState &cacheState = cacheStates[targetIdx];

	bool isNeededId = ( cacheState.cacheFromHost == 2 &&
		hashTable[targetIdx].ptr >= 0 && entriesVisibleType[targetIdx] == 0);

	if (isNeededId) shouldPrefix = true;
	__syncthreads();

	if (shouldPrefix)
	{
		int offset = computePrefixSum_device<int>(isNeededId, noNeededEntries, blockDim.x * blockDim.y, threadIdx.x);
		if (offset != -1 && offset < SDF_TRANSFER_BLOCK_NUM) neededEntryIDs[offset] = targetIdx;
	}
}

template<class TVoxel>
__global__ void cleanMemory_device(int *voxelAllocationList, int *noAllocatedVoxelEntries, ITMHashCacheState *cacheStates, 
	ITMHashEntry *hashTable, TVoxel *localVBA, int *neededEntryIDs_local, int noNeededEntries)
{
	int locId = threadIdx.x + blockIdx.x * blockDim.x;
	
	if (locId > noNeededEntries - 1) return;

	int entryDestId = neededEntryIDs_local[locId];
	
	cacheStates[entryDestId].cacheFromHost = 0;

	int vbaIdx = atomicAdd(&noAllocatedVoxelEntries[0], 1);
	if (vbaIdx < SDF_LOCAL_BLOCK_NUM - 1)
	{
		voxelAllocationList[vbaIdx + 1] = hashTable[entryDestId].ptr;
		hashTable[entryDestId].ptr = -1;
	}
}

template<class TVoxel>
__global__ void buildLocalSyncedStorageToHost_device(TVoxel *syncedVoxelBlocks_local, bool *hasSyncedData_local,
	int *neededEntryIDs_local, ITMHashEntry *hashTable, TVoxel *localVBA)
{
	int entryDestId = neededEntryIDs_local[blockIdx.x];

	ITMHashEntry &hashEntry = hashTable[entryDestId];

	TVoxel *dstVB = syncedVoxelBlocks_local + blockIdx.x * SDF_BLOCK_SIZE3;
	TVoxel *srcVB = localVBA + hashEntry.ptr * SDF_BLOCK_SIZE3;

	int vIdx = threadIdx.x + threadIdx.y * SDF_BLOCK_SIZE + threadIdx.z * SDF_BLOCK_SIZE * SDF_BLOCK_SIZE;
	dstVB[vIdx] = srcVB[vIdx];

	if (vIdx == 0) hasSyncedData_local[blockIdx.x] = true;
}

template<class TVoxel>
__global__ void integrateGlobalIntoLocal_device(TVoxel *localVBA, ITMHashCacheState *cacheStates, TVoxel *syncedVoxelBlocks_local,
	int *neededEntryIDs_local, ITMHashEntry *hashTable, int maxW)
{
	int entryDestId = neededEntryIDs_local[blockIdx.x];

	TVoxel *srcVB = syncedVoxelBlocks_local + blockIdx.x * SDF_BLOCK_SIZE3;
	TVoxel *dstVB = localVBA + hashTable[entryDestId].ptr * SDF_BLOCK_SIZE3;

	int vIdx = threadIdx.x + threadIdx.y * SDF_BLOCK_SIZE + threadIdx.z * SDF_BLOCK_SIZE * SDF_BLOCK_SIZE;

	CombineVoxelInformation<TVoxel::hasColorInformation, TVoxel>::compute(srcVB[vIdx], dstVB[vIdx], maxW);

	if (vIdx == 0) cacheStates[entryDestId].cacheFromHost = 2;
}

template<class TVoxel>
__global__ void cleanVBA(TVoxel *localVBA, int *voxelAllocationList, int baseAddressToClean)
{
	//if ((baseAddressToClean + blockIdx.x) < SDF_LOCAL_BLOCK_NUM)
	{
		int vbaIdx = voxelAllocationList[baseAddressToClean + blockIdx.x];
		TVoxel *dstVB = localVBA + vbaIdx * SDF_BLOCK_SIZE3;
		int vIdx = threadIdx.x + threadIdx.y * SDF_BLOCK_SIZE + threadIdx.z * SDF_BLOCK_SIZE * SDF_BLOCK_SIZE;
		dstVB[vIdx] = TVoxel();
	}
}

template class ITMSwappingEngine_CUDA<ITMVoxel,ITMVoxelIndex>;
